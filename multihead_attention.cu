#include "hip/hip_runtime.h"
//
// Created by CSWH on 2024/12/25.
//
#include <thread>
#include <gtest/gtest.h>
#include <utils.hpp>
#include <cub/block/block_reduce.cuh>


// 一个KV Cache + GQA的实现
// 实现这个mha的过程, 尤其是构建模拟数据的过程, 极大加深了我的理解.
// 构造测试数据是一种好的实践考验.

const int thread_num = 128;

// decoding阶段的attention, 一个block处理一个head, 一个head上的softmax
// 自回归的attention中, 一个head中有一个score, 一个head由一个block来处理. 通过在block内reduce来计算softmax
// 懒得处理out of package, 所以先不用float4
__device__ void softmax(float *input, int size) {

	float max_val = -FLT_MAX;
	// 局部最大值收集到一个block内
	for (int i = threadIdx.x; i < size; i+=blockDim.x) {
		max_val = max(max_val, input[i]);
	}
	// 最大值规约
	using BlockReduce = hipcub::BlockReduce<float, thread_num>;
	__shared__ BlockReduce::TempStorage temp;
	__shared__ float shared_val;
	max_val = BlockReduce(temp).Reduce(max_val, hipcub::Max());
	if (threadIdx.x == 0) {
		shared_val = max_val;
	}
	__syncthreads();
	max_val = shared_val;

	float sum = 0.f;
	// 局部和
	for (int i = threadIdx.x; i < size; i+=blockDim.x) {
		input[i] = expf(input[i] - max_val);

		sum += input[i];
	}
	// 规约和
	sum = BlockReduce(temp).Sum(sum);
	if (threadIdx.x == 0) {
		shared_val = sum;
	}
	__syncthreads();

	sum = shared_val;
	// 更新
	for (int i = threadIdx.x; i < size; i+=blockDim.x) {
		input[i] /= sum;
	}
}


// decoding阶段的attention, 一个block处理一个head
// 全局内存尽情用就好了, 不要怕. 先有实现才能有优化.
// 不要着急不要着急, 一点一点来, 今天写不完明天写, 今天不想做安排以后做. 不要着急. 不要老是想着一下子做完.
__global__ void mha_gpu(float *q_pos_gpu, float *k_cache_gpu, float *v_cache_gpu, float *attn_output_gpu, float *score_temp_gpu,
			int pos, int head_num, int kv_head_num, int head_dim, int max_seq) {
	int head_id = blockIdx.x;
	int head_offset = head_id * head_dim;

	int tid = threadIdx.x;
	if (head_id >= head_num) {
		return;
	}

	// head_num / kv_head_num: 一个k/v head对应多少个q head
	int kv_head_id = head_id / (head_num / kv_head_num);

	float scale = 1.0f / sqrtf(head_dim);
	// 对于当前head, q依次与各个时间点的key做内积, 计算score
	for(int t = tid; t <= pos; t+=blockDim.x) {
		int seq_offset = t*kv_head_id*head_dim;
		float* k_t = k_cache_gpu + seq_offset + head_offset;

		// 内积, 假设维度能被4整除
		float prod = 0.f;
		for(int i = 0; i < head_dim; i+=4) {
			auto& input1 = reinterpret_cast<float4 &>(q_pos_gpu[head_offset + i]);
			auto& input2 = reinterpret_cast<float4 &>(k_t[i]);
			prod += input1.x * input2.x;
			prod += input1.y * input2.y;
			prod += input1.z * input2.z;
			prod += input1.w * input2.w;
		}
		// 记录到score上
		score_temp_gpu[head_id * max_seq + t] = prod * scale; // 内积很容易变得非常大, 需要加以数值稳定调整
	}

	// 对score使用softmax
	softmax(score_temp_gpu + head_id * max_seq, pos+1);
	__syncthreads();

	// 利用score = (score_1, ..., score_pos)对v_1, ..., v_pos进行加权.
	for(int t = tid; t <= pos; t+=blockDim.x) {
		int seq_offset = t*kv_head_id*head_dim;
		float* v_t = v_cache_gpu + seq_offset + head_offset;

		float score_t = score_temp_gpu[head_id * max_seq + t];

		// score_t * v_t并铺到答案上去
		for(int i = 0; i < head_dim; i++) {
			// attn_output_gpu[head_offset + i] += score_t * v_t[i]; // 累加会出现冲突问题, 累加需要使用atomicAdd. 赋值则不需要.
			atomicAdd(&attn_output_gpu[head_offset + i], score_t * v_t[i]);
		}
	}
}
/*
 *优化:
 * 1. shared memory, 一个block需要: q_t, k_1,... ,k_t, v_1, ... ,v_t, 需求较大. 但可以内存复用, 声明一个k_t[kv_dim], v_t[kv_dim]然后循环中复用即可.
 * 2. 最后一步调整循环顺序, 减少对全局内存的访问. 但是不好理解.
 * 3. 一个block处理多个head, 太复杂, 不推荐.
 * 4. 增加float4等的使用
 * 5. 算法上的改进: flash attention
 */


TEST(test_attention, test1) {
	int head_num = 8;
	int kv_head_num = 4; // MQA
	int head_dim = 256;
	int max_seq = 512;
	int pos = 10; // 假设当前处理第10个时间点(时间点从0开始计数)

	float* q_pos_cpu = reinterpret_cast<float*>(malloc(sizeof(float) * head_num * head_dim));
	// 这里kv cache忽略了layer维度. 假设已经定位到了cache所在的layer
	float* k_cache_cpu = reinterpret_cast<float*>(malloc(sizeof(float) * max_seq * kv_head_num * head_dim));
	float* v_cache_cpu = reinterpret_cast<float*>(malloc(sizeof(float) * max_seq * kv_head_num * head_dim));

	// 假设经过了线性变换, 即q_pos = x_pos W_q
	initialize_data_random(q_pos_cpu, head_num * head_dim, 0, 2, false, 100);
	// 填充过去时间点以及当前时间点的数据, 因为线性变换后, k_pos和v_pos会被添加到cache中
	for(int seq_id = 0; seq_id < pos+1; seq_id++) {
		initialize_data_random(k_cache_cpu + seq_id * kv_head_num * head_dim, kv_head_num * head_dim, 0, 2, false, 101);
		initialize_data_random(v_cache_cpu + seq_id * kv_head_num * head_dim, kv_head_num * head_dim, 0, 2, false, 102);
	}

	float* q_pos_gpu, * k_cache_gpu, * v_cache_gpu, * attn_output_gpu, * score_temp_gpu;
	hipMalloc(&q_pos_gpu, head_num * head_dim * sizeof(float));
	hipMalloc(&k_cache_gpu, max_seq * kv_head_num * head_dim * sizeof(float));
	hipMalloc(&v_cache_gpu, max_seq * kv_head_num * head_dim * sizeof(float));
	hipMalloc(&score_temp_gpu, head_num * max_seq * sizeof(float)); // 给所有时间点都准备了权重.
	hipMalloc(&attn_output_gpu, head_num * head_dim * sizeof(float)); // 输出是跟q一样大的, 实际中可以直接复用

	hipMemcpy(q_pos_gpu, q_pos_cpu, head_num * head_dim * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(k_cache_gpu, k_cache_cpu, kv_head_num * max_seq * head_dim * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(v_cache_gpu, v_cache_cpu, kv_head_num * max_seq * head_dim * sizeof(float), hipMemcpyHostToDevice);

	mha_gpu<<<head_num, thread_num>>>(q_pos_gpu, k_cache_gpu, v_cache_gpu, attn_output_gpu, score_temp_gpu,
			pos, head_num, kv_head_num, head_dim, max_seq);

	float* output_cpu = reinterpret_cast<float*>(malloc(sizeof(float) * head_num * head_dim));
	hipMemcpy(output_cpu, attn_output_gpu, head_num * head_dim * sizeof(float), hipMemcpyDeviceToHost);

	printf("q_pos:\n");
	show_matrix(q_pos_cpu, 1, 100);// 第一个头的前100
	printf("result\n");
	show_matrix(output_cpu, 1, 100);

	hipError_t err = hipGetLastError(); // 检查是否有CUDA错误
	if (err != hipSuccess) {
		printf("CUDA error: %s\n", hipGetErrorString(err));
	}

	free(q_pos_cpu);
	free(k_cache_cpu);
	free(v_cache_cpu);
	free(output_cpu);

	hipFree(q_pos_gpu);
	hipFree(k_cache_gpu);
	hipFree(v_cache_gpu);
	hipFree(attn_output_gpu);
	hipFree(score_temp_gpu);
}
